#include<stdio.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>

__global__ void duplicate(char* sin, char *sout,int len)
{
  int tid = threadIdx.x;
  int n = blockDim.x;
  int si = tid*len;
  for(int i=0;i<len;i++)
  {
    sout[si++] = sin[i];
  }
}

int main()
{
  char * sin, * sout, *d_sin, * d_sout;
  int len,n,soutlen;

  
  printf("Enter number of times to repeat the string: ");
  scanf("%d",&n);
  sin = (char*) malloc(sizeof(char)*20);
  sout = (char*) malloc(sizeof(char)*20*n);
  printf("Enter string:");
  scanf("%s",sin);
  len = strlen(sin);
  soutlen = len*n;

  hipMalloc((void**)&d_sin,sizeof(char)*len);
  hipMalloc((void**)&d_sout,sizeof(char)*soutlen);
  hipMemcpy(d_sin,sin,sizeof(char)*len,hipMemcpyHostToDevice);

  duplicate<<<1,n>>>(d_sin,d_sout,len);
  hipMemcpy(sout,d_sout,sizeof(char)*soutlen,hipMemcpyDeviceToHost);

  printf("Modified String: %s",sout);

}