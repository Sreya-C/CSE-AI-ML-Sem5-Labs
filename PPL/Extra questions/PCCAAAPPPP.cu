#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdlib.h>

__global__ void modify(char * str, char * rstr)
{
  int tid = threadIdx.x,currpos=0;

  for(int i=0;i<=tid;i++)
  {
    currpos += i;
  }
  for(int i=0;i<tid+1;i++)
  {
    rstr[currpos++] = str[tid]; 
  }
}

int main()
{
  char * str, * d_str, * rstr, * d_rstr;
  int len;

  str = (char * ) malloc(sizeof(char)*30);
  rstr = (char * ) malloc(sizeof(char)*60);
  printf("Enter a string: ");
  scanf("%s",str);
  len = strlen(str);
  hipMalloc((void**)&d_str,sizeof(char)*len);
  hipMalloc((void**)&d_rstr,sizeof(char)*len*(len+1)/2);
  hipMemcpy(d_str,str,sizeof(char)*len,hipMemcpyHostToDevice);

  modify<<<1,len>>>(d_str,d_rstr);

  hipMemcpy(rstr,d_rstr,sizeof(char)*(len+1)*len/2,hipMemcpyDeviceToHost);

  printf("Modified string: %s",rstr);
}