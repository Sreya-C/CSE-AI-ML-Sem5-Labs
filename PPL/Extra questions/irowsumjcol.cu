#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void rowsum(int * A,int col, int * rowsums)
{
  int rid = threadIdx.x,sum=0;
  for(int i=0;i<col;i++)
  {
    sum += A[rid*col+i];
  }
  rowsums[rid] = sum;
}

__global__ void colsum(int * A,int row, int * colsums)
{
  int cid = threadIdx.x,sum=0;
  int col = blockDim.x;
  for(int i=0;i<row;i++)
  {
    sum += A[i*col+cid];
  }
  colsums[cid] = sum;
}

__global__ void replacemat(int * rowsums, int * colsums,int *B)
{
  int rid = threadIdx.y,cid = threadIdx.x;
  int col = blockDim.x;
  B[rid*col + cid] = rowsums[rid]+colsums[cid];
}

int main()
{
  int *A,*rowsums,*colsums,row,col,matsize,*B;
  int *d_A,*d_rowsums,*d_colsums,*d_B;
  printf("Enter rowsize and colsize of matrix: ");
  scanf("%d %d",&row,&col);
  printf("%d %d",row,col);
  matsize = row*col*sizeof(int);
  A = (int *) malloc(matsize);
  B = (int *) malloc(matsize);
  rowsums = (int *) malloc(sizeof(int)*row);
  colsums = (int *) malloc(sizeof(int)*col);
  printf("Enter elements of matrix: \n");
  for(int i=0;i<row*col;i++)
  {
    scanf("%d",&A[i]);
  }
  for(int i=0;i<row;i++)
  {
    for(int j=0;j<col;j++)
    {
      printf("%d",A[i*col+j]);
    }
    printf("\n");
  }
  hipMalloc((void**)&d_A,matsize);
  hipMalloc((void**)&d_B,matsize);
  hipMalloc((void**)&d_rowsums,sizeof(int)*row);
  hipMalloc((void**)&d_colsums,sizeof(int)*col);
  hipMemcpy(d_A,A,matsize,hipMemcpyHostToDevice);

  rowsum<<<1,row>>>(d_A,col,d_rowsums);
  colsum<<<1,col>>>(d_A,row,d_colsums);
  hipDeviceSynchronize();
  hipMemcpy(rowsums,d_rowsums,sizeof(int)*row,hipMemcpyDeviceToHost);
  hipMemcpy(colsums,d_colsums,matsize/row,hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  printf("ROWSUMS: \n");
  for(int i=0;i<row;i++)
  {
    printf("%d ",rowsums[i]);
  }
  printf("\nCOLSUMS: \n");  
  for(int i=0;i<col;i++)
  {
    printf("%d ",colsums[i]);
  }

  dim3 dimGrid(1,1,1);
  dim3 dimBlock(col,row,1);
  replacemat<<<dimGrid,dimBlock>>>(d_rowsums,d_colsums,d_B);

  hipMemcpy(B,d_B,matsize,hipMemcpyDeviceToHost);
  printf("\nModified matrix: \n");
  for(int i=0;i<row;i++)
  {
    for(int j=0;j<col;j++)
    {
      printf("%d ",B[i*col+j]);
    }
    printf("\n");
  }
}
