/*Write a CUDA program that reads a MXN matrix A and produces a resultant matrix B of 
same size as follows: Replace all the even numbered matrix elements with their row sum 
and odd numbered matrix elements with their column sum.
Example: A B
I/p: 1 2 3 O/p: 5 6 9
4 5 6 15 7 15*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void rowsum(int * A,int col,int * rowsums)
{
  int tid = threadIdx.x;
  int sum = 0;
  for(int i=0;i<col;i++)
  {
    sum += A[col*tid+i];
  }
  rowsums[tid] = sum;
  printf("At Row %d, rowsum = %d\n",tid,rowsums[tid]);
}

__global__ void colsum(int * A,int row,int * colsums)
{
  int tid = threadIdx.x,col = blockDim.x,sum = 0;
  colsums[tid] = 0;
  printf("Calculating colsum: ROWS %d,COLS %d,TID: %d\n",row,col,tid);
  for(int i=0;i<row;i++)
  {
    sum += A[i*col+tid];
    printf("%d,%d,%d\n",tid,i*col+tid,A[i*col+tid]);
  }
  colsums[tid] = sum;
  printf("At Col %d, colsum = %d\n",tid,colsums[tid]);
}

__global__ void replacemat(int * A, int * rowsums, int * colsums)
{
  int rid = threadIdx.y,cid = threadIdx.x;
  int row = blockDim.y,col = blockDim.x;
  int ele = A[rid*col+cid];
  if (ele%2==0) A[rid*col + cid] = rowsums[rid];
  else A[rid*col + cid] = colsums[cid];
}

int main()
{
  int *A,*rowsums,*colsums,row,col,matsize;
  int *d_A,*d_rowsums,*d_colsums;
  printf("Enter rowsize and colsize of matrix: ");
  scanf("%d %d",&row,&col);
  printf("%d %d",row,col);
  matsize = row*col*sizeof(int);
  A = (int *) malloc(matsize);
  rowsums = (int *) malloc(sizeof(int)*row);
  colsums = (int *) malloc(sizeof(int)*col);
  printf("Enter elements of matrix: \n");
  for(int i=0;i<row*col;i++)
  {
    scanf("%d",&A[i]);
  }
  for(int i=0;i<row;i++)
  {
    for(int j=0;j<col;j++)
    {
      printf("%d",A[i*col+j]);
    }
    printf("\n");
  }
  hipMalloc((void**)&d_A,matsize);
  hipMalloc((void**)&d_rowsums,sizeof(int)*row);
  hipMalloc((void**)&d_colsums,sizeof(int)*col);
  hipMemcpy(d_A,A,matsize,hipMemcpyHostToDevice);

  rowsum<<<1,row>>>(d_A,col,d_rowsums);
  colsum<<<1,col>>>(d_A,row,d_colsums);
  hipDeviceSynchronize();
  hipMemcpy(rowsums,d_rowsums,sizeof(int)*row,hipMemcpyDeviceToHost);
  hipMemcpy(colsums,d_colsums,matsize/row,hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  printf("ROWSUMS: \n");
  for(int i=0;i<row;i++)
  {
    printf("%d ",rowsums[i]);
  }
  printf("\nCOLSUMS: \n");  
  for(int i=0;i<col;i++)
  {
    printf("%d ",colsums[i]);
  }

  dim3 dimGrid(1,1,1);
  dim3 dimBlock(col,row,1);
  replacemat<<<dimGrid,dimBlock>>>(d_A,d_rowsums,d_colsums);

  hipMemcpy(A,d_A,matsize,hipMemcpyDeviceToHost);
  printf("\nModified matrix: \n");
  for(int i=0;i<row;i++)
  {
    for(int j=0;j<col;j++)
    {
      printf("%d ",A[i*col+j]);
    }
    printf("\n");
  }
}