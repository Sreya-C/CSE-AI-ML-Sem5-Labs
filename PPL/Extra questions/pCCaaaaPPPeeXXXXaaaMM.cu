#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string.h>

__global__ void corr(char *A,int*cumul,int *B,char * rstr)
{
  int rid = threadIdx.y,cid = threadIdx.x;
  int col = blockDim.x;
  int si = (cumul[rid*col+cid]-B[rid*col+cid])> 0 ? cumul[rid*col+cid -1] : 0;
  int ei = cumul[rid*col+cid];
  for(int i=si;i<ei;i++)
  {
    rstr[i] = A[rid*col+cid];
  }
}

int main()
{
  char *A,*rstr,*d_rstr,*d_A;
  int *B,*cumul,*d_cumul,*d_B,row,col,size;
  printf("Enter matrix dimensions: ");
  scanf("%d %d",&row,&col);
  size = row*col*sizeof(int);
  A = (char*) malloc(sizeof(char)*row*col);
  B = (int*) malloc(size);
  cumul = (int*) malloc(size);
  fflush(stdin);
  printf("Enter character matrix:");
  fflush(stdin);
  scanf("%c",&A[0]);
  for(int i=0;i<row;i++)
  {
    for(int j=0;j<col;j++)
    {
      scanf("%c",&A[i*col+j]);
      fflush(stdin);
    }
  }
  fflush(stdin);
  printf("Character Matrix: \n");
  for(int i=0;i<row;i++)
  {
    for(int j=0;j<col;j++)
    {
      printf("%c ",A[i*col+j]);
    }
    printf("\n");
  }
  printf("Enter number matrix: \n");
  int sum = 0;
  for(int i=0;i<row;i++)
  {
    for(int j=0;j<col;j++)
    {
      scanf("%d",&B[i*col+j]);
      sum += B[i*col+j];
      cumul[i*col+j] = sum;
    }
  }
  printf("Number Matrix: \n");
  for(int i=0;i<row;i++)
  {
    for(int j=0;j<col;j++)
    {
      printf("%d",B[i*col+j]);
    }
    printf("\n");
  }
  printf("Cumulative Sums Matrix: \n");
  for(int i=0;i<row;i++)
  {
    for(int j=0;j<col;j++)
    {
      printf("%d ",cumul[i*col+j]);
    }
    printf("\n");
  }
  int rstrlen = cumul[row*col-1];
  rstr = (char*) malloc(sizeof(char)*(rstrlen));
  hipMalloc((void**)&d_A,sizeof(char)*row*col);
  hipMalloc((void**)&d_B,size);
  hipMalloc((void**)&d_cumul,size);
  hipMalloc((void**)&d_rstr,sizeof(char)*(rstrlen));
  hipMemcpy(d_A,A,sizeof(char)*row*col,hipMemcpyHostToDevice);
  hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);
  hipMemcpy(d_cumul,cumul,size,hipMemcpyHostToDevice);
  dim3 dimGrid(1,1,1);
  dim3 dimBlock(col,row,1);
  corr<<<dimGrid,dimBlock>>>(d_A,d_cumul,d_B,d_rstr);
  hipMemcpy(rstr,d_rstr,sizeof(char)*(rstrlen),hipMemcpyDeviceToHost);

  printf("Formed String: %s",rstr);
}