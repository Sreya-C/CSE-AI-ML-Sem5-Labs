#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdlib.h>

__global__ void reversen(char * str, int * si)
{
  int tid = threadIdx.x;
  printf("\nTID: %d, starting index of tid %d",tid,si[tid]);
  int start_ind = si[tid];
  int end_ind = si[tid+1] - 1;
  int len = (end_ind - start_ind)/2 ;
  if(len==1) return;
  printf("TID: %d, Start Index: %d, End Index: %d, Length = %d\n",tid,start_ind,end_ind,len);
  for (int i = 0; i < len; i++) 
  { 
    int temp = str[start_ind + i]; 
    str[start_ind + i] = str[end_ind - i - 1];
    str[end_ind - i - 1] = temp;
  }
}

int main()
{
  char *str, * d_str;
  int len,ind=0,i;
  int *si,*d_si;
  str = (char * ) malloc(sizeof(char)*50);
  si = (int * ) malloc(sizeof(int)*20);
  printf("Enter a string of words: ");
  scanf("%[^\n]s",str);
  len = strlen(str);
  si[ind++] = 0;
  for(i=0;i<len;i++)
  {
    if(str[i] == ' ')
    {
      si[ind++] = i+1;
    }
  }
  si[ind] = i+1;
  int numWords = ind;
  for(int j=0;j<numWords+2;j++)
  {
    printf("%d ",si[j]);
  }
  hipMalloc((void**)&d_str,sizeof(char)*len);
  hipMalloc((void**)&d_si,sizeof(int)*(numWords+1));
  hipMemcpy(d_str,str,sizeof(char)*len,hipMemcpyHostToDevice);
  hipMemcpy(d_si,si,sizeof(int)*(numWords+1),hipMemcpyHostToDevice);

  reversen<<<1,numWords>>>(d_str,d_si);
  hipMemcpy(str,d_str,sizeof(char)*(len),hipMemcpyDeviceToHost);

  printf("Modified string: %s",str);
}

