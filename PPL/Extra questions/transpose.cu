#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

__global__ void transpose(int * mat, int *rmat)
{
  int rid = threadIdx.y;
  int cid = threadIdx.x;
  int row = blockDim.y,col = blockDim.x;
  rmat[cid*row+rid] = mat[rid*col+cid];
}

int main()
{
  int *mat,*d_mat,*rmat,*d_rmat;
  int row,col;
  printf("Enter row and column size: ");
  scanf("%d %d",&row,&col);
  mat = (int *) malloc(row*col*sizeof(int));
  rmat = (int *) malloc(row*col*sizeof(int));
  printf("Enter %dx%d matrix: \n",row,col);
  for(int i=0;i<row*col;i++)
  {
    scanf("%d",&mat[i]);
  }
  hipMalloc((void**)&d_mat,sizeof(int)*row*col);
  hipMalloc((void**)&d_rmat,sizeof(int)*row*col);
  hipMemcpy(d_mat,mat,sizeof(int)*row*col,hipMemcpyHostToDevice);

  dim3 dimGrid(1,1,1);
  dim3 dimBlock(col,row,1);

  transpose<<<dimGrid,dimBlock>>>(d_mat,d_rmat);
  hipMemcpy(rmat,d_rmat,sizeof(int)*row*col,hipMemcpyDeviceToHost);

  for(int i=0;i<col;i++)
  {
    for(int j=0;j<row;j++)
    {
      printf("%d",rmat[i*row+j]);
    }
    printf("\n");
  }

}