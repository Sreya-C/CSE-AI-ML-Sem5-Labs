#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


#define MAX_MASK_WIDTH 10
__constant__ float M[MAX_MASK_WIDTH];

__global__ void convolve(float* N, float* P, int width, int mask_width) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(tid < width) {
		int start = tid - mask_width / 2;
		P[tid] = 0.0;
		for(int i=0; i<mask_width; i++) 
			if(start + i >= 0 && start + i < width)
				P[tid] += N[start + i] * M[i];
	}		
}

int main(void) {
	int width, mask_width, size_N, size_M;
	float *h_N, *h_M, *h_P;
	float *d_N, *d_P;
	
	printf("Enter the length of the input array N: ");
	scanf("%d", &width);
	
	size_N = sizeof(float) * width;
	h_N = (float*) malloc(size_N);
	
	printf("Enter the input array N: ");
	for(int i=0; i<width; i++) 
		scanf("%f", h_N+i);
	
	printf("Enter the length of the mask array M: ");
	scanf("%d", &mask_width);
	
	if(mask_width % 2 == 0) {
		printf("Mask length should be odd.\n");
		free(h_N);
		exit(1);
	}
	
	size_M = sizeof(float) * mask_width;
	h_M = (float*) malloc(size_M);
	
	printf("Enter the mask array M: ");
	for(int i=0; i<mask_width; i++) 
		scanf("%f", h_M+i);
		
	h_P = (float*) malloc(size_N);
	
	hipMalloc((void**)&d_N, size_N);
	hipMalloc((void**)&d_P, size_N);
	
	hipMemcpy(d_N, h_N, size_N, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(M), h_M, size_M);	
	convolve<<<ceil(width/256.0), 256>>>(d_N, d_P, width, mask_width);
	
	hipMemcpy(h_P, d_P, size_N, hipMemcpyDeviceToHost);
	
	printf("Result of convolution, output array P: ");
	for(int i=0; i<width; i++) 
		printf("%.2f ", h_P[i]);
	printf("\n");
	
	hipFree(d_N);
	hipFree(d_P);
	
	free(h_N);
	free(h_M);
	free(h_P);
	
	return 0;
}	