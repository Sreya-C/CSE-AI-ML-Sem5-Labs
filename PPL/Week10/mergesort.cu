#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


__device__ void merge_sequential(int* A, int m, int* B, int n, int* C) {
    int i=0, j=0, k=0;

    while(i<m && j<n) 
        if(A[i] <= B[j]) 
            C[k++] = A[i++];
        else 
            C[k++] = B[j++];
    
    if(i == m) 
        while(j < n) 
            C[k++] = B[j++];
    else 
        while(i < m) 
            C[k++] = A[i++];
}

__device__ int co_rank(int k, int* A, int m, int* B, int n) {
    int i = k < m ? k : m;
    int j = k - i;
    int i_low = 0 > (k-n) ? 0 : k-n;
    int j_low = 0 > (k-m) ? 0 : k-m;
    int delta;
    int active = 1;

    while(active) {
        if(i > 0 && j < n && A[i-1] > B[j]) {
            delta = ((i - i_low + 1) >> 1);
            j_low = j;
            j = j + delta;
            i = i - delta;
        }
        else if(j > 0 && i < m && B[j-1] >= A[i]) {
            delta = ((j - j_low + 1) >> 1);
            i_low = i;
            i = i + delta;
            j = j - delta;
        }
        else {
            active = 0;
        }
    }

    return i;
}

__global__ void merge_kernel(int* A, int m, int* B, int n, int* C) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int elementsPerThread = ceil((double)(m+n)/(blockDim.x*gridDim.x));
    int k_curr = tid * elementsPerThread;
    int k_next = min((tid + 1) * elementsPerThread, m+n);
    int i_curr = co_rank(k_curr, A, m, B, n);
    int i_next = co_rank(k_next, A, m, B, n);
    int j_curr = k_curr - i_curr;
    int j_next = k_next - i_next;

    merge_sequential(&A[i_curr], i_next-i_curr, &B[j_curr], j_next-j_curr, &C[k_curr]);
}

int main() {
    int *A, *B, *C;
    int m, n, sizeA, sizeB, sizeC;
    int *d_A, *d_B, *d_C;

    printf("Enter the size of the first array, A: ");
    scanf("%d", &m);

    printf("Enter the size of the second array, B: ");
    scanf("%d", &n);

    sizeA = sizeof(int) * m;
    sizeB = sizeof(int) * n;
    sizeC = sizeof(int) * (m+n);

    A = (int*) malloc(sizeA);
    B = (int*) malloc(sizeB);
    C = (int*) malloc(sizeC);

    printf("Enter the sorted array, A: ");
    for(int i=0; i<m; i++) 
        scanf("%d", A+i);

    printf("Enter the sorted array, B: ");
    for(int i=0; i<n; i++) 
        scanf("%d", B+i);

    hipMalloc((void**) &d_A, sizeA);
    hipMalloc((void**) &d_B, sizeB);
    hipMalloc((void**) &d_C, sizeC);
    hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);

    merge_kernel<<<1, ceil((m+n)/5.0)>>>(d_A, m, d_B, n, d_C);

    hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);

    printf("Resultant Array after Parallel Merge Sorting:\n");
    for(int i=0; i<m+n; i++) 
        printf("%4d", C[i]);
    printf("\n");

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    free(A);
    free(B);
    free(C);
    
    return 0;
}