#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>

__global__ void oddKernel(int * A)
{
	int tid = threadIdx.x;
	if((tid%2!=0) && tid < (blockDim.x - 1))
	{
		if(A[tid]>A[tid+1])
		{
			int temp = A[tid];
			A[tid] = A[tid+1];
			A[tid+1] = temp;
		}
	}
}

__global__ void evenKernel(int * A)
{
	int tid = threadIdx.x;
	if((tid%2==0) && tid < (blockDim.x - 1))
	{
		if(A[tid]>A[tid+1])
		{
			int temp = A[tid];
			A[tid] = A[tid+1];
			A[tid+1] = temp;
		}
	}
}

int main()
{
	int size;
	printf("Enter size of array: ");
	scanf("%d",&size);
	int A[size];
	printf("Enter %d elements of array: ",size);
	for(int i=0;i<size;i++) scanf("%d",&A[i]);
	int * d_A;
	
	hipMalloc((void**)&d_A,sizeof(int)*size);
	hipMemcpy(d_A,A,sizeof(int)*size,hipMemcpyHostToDevice);
	for(int i=0;i<=size/2;i++)
	{
		oddKernel<<<1,size>>>(d_A);
		evenKernel<<<1,size>>>(d_A);
	}
	hipMemcpy(A,d_A,sizeof(int)*size,hipMemcpyDeviceToHost);
	printf("Sorted Array: ");
	for(int i=0;i<size;i++) printf("%d ",A[i]);
}
