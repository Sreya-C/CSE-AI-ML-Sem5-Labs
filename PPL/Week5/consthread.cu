#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define N 10

__global__ void vecAdd(float *a, float *b, float *c) {
  int tid = threadIdx.x + blockIdx.x*blockDim.x;
  if (tid < N) {
    printf("%d,%d,%d,%d\n",threadIdx.x,blockDim.x,blockIdx.x,gridDim.x);
    c[tid] = a[tid] + b[tid];
  }
}

int main() {
  float *h_a = (float *)malloc(N * sizeof(float));
  float *h_b = (float *)malloc(N * sizeof(float));
  float *h_c = (float *)malloc(N * sizeof(float));
  for (int i = 0; i < N; i++) {
    h_a[i] = i;
    h_b[i] = 3*i;
  }

  float *d_a, *d_b, *d_c;
  hipMalloc(&d_a, N * sizeof(float));
  hipMalloc(&d_b, N * sizeof(float));
  hipMalloc(&d_c, N * sizeof(float));
  hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);

  int numblocks = ceil(N/256)+1;
  vecAdd<<<numblocks,256>>>(d_a, d_b, d_c);
  hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);
  for (int i = 0; i < N; i++) {
    printf("%f\t", h_c[i]);
  }

  free(h_a);
  free(h_b);
  free(h_c);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  return 0;
}
