#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void conv1D(float * N,float * M,float * P,float width,float mask_width)
{
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	int pvalue = 0,j,start_point = tid-(mask_width/2);
	for(j=0;j<mask_width;j++)
	{
		if(start_point+j >=0 && start_point+j<width)
			pvalue += N[start_point+j]*M[j];
	}
	P[tid] = pvalue;

}	


int main()
{	
	int width,mask_width;
	printf("Enter array width and mask width");
	scanf("%d",&width);
	scanf("%d",&mask_width);
	int arrsize = width*sizeof(float);
	int masksize = mask_width*sizeof(float);
	float *h_N = (float *)malloc(arrsize);
  	float *h_M = (float *)malloc(masksize);
  	float *h_P = (float *)malloc(arrsize);

  	float *d_N, *d_M, *d_P;
	hipMalloc(&d_N, arrsize);
	hipMalloc(&d_M, masksize);
	hipMalloc(&d_P, arrsize);
	printf("Enter elements of array:");
	for(int i=0;i<width;i++) scanf("%f",h_N + i);
	hipMemcpy(d_N,h_N,arrsize,hipMemcpyHostToDevice);
	printf("Enter elements of mask:");
	for(int i=0;i<mask_width;i++) scanf("%f",h_M + i);
	hipMemcpy(d_M,h_M,masksize,hipMemcpyHostToDevice);

	conv1D<<<ceil(width/32),32>>>(d_N,d_M,d_P,width,mask_width);
	hipMemcpy(h_P,d_P,arrsize,hipMemcpyDeviceToHost);

	printf("Array: ");
	for(int i=0;i<width;i++)
	{
		printf("%d ",(int)h_N[i]);
	}

	printf("\nMask Array: ");
	for(int i=0;i<mask_width;i++)
	{
		printf("%d ",(int)h_M[i]);
	}

	printf("\nConvolved Array: ");
	for(int i=0;i<width;i++)
	{
		printf("%d ",(int)h_P[i]);
	}

	free(h_N);
	free(h_M);
	free(h_P);

	hipFree(d_N);
	hipFree(d_M);
	hipFree(d_P);
}