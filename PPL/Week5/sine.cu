#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define N 15

__global__ void findsin(float * a,float * output)
{
    int tid = blockDim.x*blockIdx.x + threadIdx.x;

    if(tid<N){
        output[tid] = sin(a[tid]);
    }
}

int main()
{
  float *h_a = (float *)malloc(N * sizeof(float));
  float *output = (float *)malloc(N * sizeof(float));
  for (int i = 0; i < N; i++) {
    h_a[i] = i*0.3;
  }

  float *d_a, *d_output;
  hipMalloc(&d_a, N * sizeof(float));
  hipMalloc(&d_output, N * sizeof(float));
  hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);

  findsin<<<1, N>>>(d_a,d_output);
  hipMemcpy(output, d_output, N * sizeof(float), hipMemcpyDeviceToHost);
  printf("Using N blocks\n");
  for (int i = 0; i < N; i++) {
    printf("%f\n", output[i]);
  }

}