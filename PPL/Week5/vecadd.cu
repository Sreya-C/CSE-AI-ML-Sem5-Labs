#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 5

__global__ void vecAdd1(float *a, float *b, float *c) {
  int id = threadIdx.x;
  if (id < N) {
    c[id] = a[id] + b[id];
  }
}
__global__ void vecAdd2(float *a, float *b, float *c) {
  int id = blockIdx.x;
  if (id < N) {
    c[id] = a[id] + b[id];
  }
}

int main() {
  float *h_a = (float *)malloc(N * sizeof(float));
  float *h_b = (float *)malloc(N * sizeof(float));
  float *h_c = (float *)malloc(N * sizeof(float));
  for (int i = 0; i < N; i++) {
    h_a[i] = i;
    h_b[i] = i+3;
  }

  float *d_a, *d_b, *d_c;
  hipMalloc(&d_a, N * sizeof(float));
  hipMalloc(&d_b, N * sizeof(float));
  hipMalloc(&d_c, N * sizeof(float));
  hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);

  vecAdd1<<<1, N>>>(d_a, d_b, d_c);
  hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);
  printf("Using N blocks\n");
  for (int i = 0; i < N; i++) {
    printf("%f\n", h_c[i]);
  }

  vecAdd2<<<N, 1>>>(d_a, d_b, d_c);
  hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);
  printf("Using N threads\n");
  for (int i = 0; i < N; i++) {
    printf("%f\n", h_c[i]);
  }

  free(h_a);
  free(h_b);
  free(h_c);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  return 0;
}
