#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#define N 1024

__global__ void CUDACount(char* A, unsigned int *d_count){
	int i = threadIdx.x;
	if(A[i]=='a')
		atomicAdd(d_count,1);
}

int main() 
{
	char A[N];
	char *d_A;
	unsigned int count=0,*d_count,result=0;
	printf("Enter a string: ");
	scanf("%s[^/n]",A);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	hipMalloc((void**)&d_A, strlen(A)*sizeof(char));
	hipMalloc((void**)&d_count,sizeof(unsigned int));
	hipMemcpy(d_A, A, strlen(A)*sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(d_count,&count,sizeof(unsigned int),hipMemcpyHostToDevice);
	hipError_t error =hipGetLastError();
	if (error != hipSuccess) {
		printf("CUDA Error1: %s\n", hipGetErrorString(error));
	}
	CUDACount<<<1,strlen(A)>>>(d_A,d_count);
	error =hipGetLastError();
	if (error != hipSuccess) {
		printf("CUDA Error2: %s\n", hipGetErrorString(error));
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipMemcpy(&result, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost);
	printf("Total occurences of a= %u\n",result);
	printf("Time Taken=%f\n",elapsedTime);
	hipFree(d_A);
	hipFree(d_count);
	printf("\n");
	return 0;
}

