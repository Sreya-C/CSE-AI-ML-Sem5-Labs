#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include<string.h>

#define N 1024
__global__ void wordcount(char *A,char* B,int n, unsigned int *d_count){
    int i = threadIdx.x;
      for(int j=0; j<n; j++){
        if(A[i+j] != B[j]) 
            return;}
    atomicAdd(d_count, 1);
}

int main() 
{
    char A[N],B[N];
    char *d_A,*d_B;

    unsigned int *count,*result,*d_count;
    count=(unsigned int *) malloc(1 * sizeof(unsigned int));
    result=(unsigned int *) malloc(1 * sizeof(unsigned int));

    printf("Enter a string:");
    scanf("%s",A);
    printf("ENTER THE WORD:");
    scanf("%s",B);


    hipMalloc((void**)&d_A, strlen(A)*sizeof(char));
    hipMalloc((void**)&d_B, strlen(B)*sizeof(char));
    hipMalloc((void **)&d_count,sizeof(unsigned int));
    hipMemcpy(d_A, A, strlen(A)*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, strlen(B)*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_count,count,sizeof(unsigned int),hipMemcpyHostToDevice);

    wordcount<<<1, strlen(A)-strlen(B)+1>>>(d_A, d_B, strlen(B), d_count);
    
    hipMemcpy(result, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost);
    printf("Total occurences of %s=%u\n",B,*result);
    hipFree(d_A);
    hipFree(d_count);
    printf("\n");
    return 0;
}