#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void matrixaddrow(int * A, int * B, int * C, int col)
{
	int rid = threadIdx.x;
	int row = blockDim.x;
	if(rid<row)
	{
		for(int cid=0;cid<col;cid++)
		{	
			C[rid*col+cid] = A[rid*col+cid] + B[rid*col+cid];
		}
	}
}

__global__ void matrixaddcol(int * A, int * B, int * C, int row)
{
	int cid = threadIdx.x;
	int col = blockDim.x;
	if(cid<col)
	{
		for(int rid = 0;rid<row;rid++)
		{
			C[rid*col+cid] = A[rid*col+cid] + B[rid*col+cid];
		}
	}
}

__global__ void matrixaddrc(int * A, int * B, int * C)
{
	int cid = threadIdx.x, col = blockDim.x;
	int rid = threadIdx.y, row = blockDim.y;
	if(cid<col && rid<row)
	{
		C[rid*col+cid] = A[rid*col+cid] + B[rid*col+cid];
	}
}


int main()
{
	int A[100][100],linearA[500],linearB[500],B[100][100],C[100][100],linearC[500];
	int *d_A,*d_B,*d_C,row,col,matsize,index = 0;
	printf("Enter num of rows and num of columns of matrices: ");
	scanf("%d %d",&row,&col);
	matsize = row*col*sizeof(int);
	printf("Enter matrix A of size %dx%d:\n",row,col);
	for(int i=0;i<row;i++){
		for(int j=0;j<col;j++){
			scanf("%d",&A[i][j]);
			linearA[index++] = A[i][j];
		}
	}
	printf("Enter matrix B of size %dx%d:\n",row,col);
	index = 0;
	for(int i=0;i<row;i++){
		for(int j=0;j<col;j++){
			scanf("%d",&B[i][j]);
			linearB[index++] = B[i][j];
		}
	}
	hipMalloc((void**)&d_A,matsize);
	hipMalloc((void**)&d_B,matsize);
	hipMalloc((void**)&d_C,matsize);
	hipMemcpy(d_A,linearA,matsize,hipMemcpyHostToDevice);
	hipMemcpy(d_B,linearB,matsize,hipMemcpyHostToDevice);

	matrixaddrow<<<1,row>>>(d_A,d_B,d_C,col);

	printf("Each row in a separate thread: \n");
	hipMemcpy(linearC,d_C,matsize,hipMemcpyDeviceToHost);
	index = 0;
	for(int i=0;i<row;i++){
		for(int j=0;j<col;j++){
			C[i][j] = linearC[index++];
		}
	}
	printf("Sum of the 2 matrices:\n");
	for(int i=0;i<row;i++){
		for(int j=0;j<col;j++){
			printf("%d ",C[i][j]);
		}
		printf("\n");
	}

	matrixaddcol<<<1,col>>>(d_A,d_B,d_C,row);
	printf("Each col in a separate thread: \n");
	hipMemcpy(linearC,d_C,matsize,hipMemcpyDeviceToHost);
	index = 0;
	for(int i=0;i<row;i++){
		for(int j=0;j<col;j++){
			C[i][j] = linearC[index];
			index += 1;
		}
	}
	printf("Sum of the 2 matrices:\n");
	for(int i=0;i<row;i++){
		for(int j=0;j<col;j++){
			printf("%d ",C[i][j]);
		}
		printf("\n");
	}

	dim3 dimGrid(1,1,1);
	dim3 dimBlock(col,row,1);
	matrixaddrc<<<dimGrid,dimBlock>>>(d_A,d_B,d_C);
	printf("Each element in a separate thread: \n");
	hipMemcpy(linearC,d_C,matsize,hipMemcpyDeviceToHost);
	index = 0;
	for(int i=0;i<row;i++){
		for(int j=0;j<col;j++){
			C[i][j] = linearC[index++];
		}
	}
	printf("Sum of the 2 matrices:\n");
	for(int i=0;i<row;i++){
		for(int j=0;j<col;j++){
			printf("%d ",C[i][j]);
		}
		printf("\n");
	}
	hipFree(d_A);hipFree(d_B);hipFree(d_C);
}