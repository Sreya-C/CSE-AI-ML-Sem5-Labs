#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void matrixmulrow(int * A, int * B, int * C, int WB,int WA)
{
	int ridA = threadIdx.x, HA = blockDim.x, sum = 0;
	if(ridA<HA)
	{
		for(int cidB = 0;cidB<WB;cidB++)
		{
			sum = 0;
			for(int k=0;k<WA;k++)
			{
				printf("%d,%d\n",A[ridA*WA + k],B[k*WB+cidB]);
				sum += A[ridA*WA + k]*B[k*WB+cidB];
			}
			printf("Sum = %d",sum);
			C[ridA*WB + cidB] = sum;
		}
	}
}

__global__ void matrixmulcol(int * A, int * B, int * C, int HA, int WA)
{
	int cidB = threadIdx.x, WB = blockDim.x, sum = 0;
	if(cidB<WB)
	{
		for(int ridA = 0;ridA<HA;ridA++)
		{
			sum = 0;
			for(int k=0;k<WA;k++)
			{
				sum += A[ridA*WA + k]*B[k*WB+cidB];
			}
			C[ridA*WB + cidB] = sum;
		}
	}
}

__global__ void matrixaddrc(int * A, int * B, int * C, int WA)
{
	int cidB = threadIdx.x, WB = blockDim.x;
	int ridA = threadIdx.y, HA = blockDim.y;
	int sum = 0;
	if(cidB<WB && ridA<HA)
	{
		for(int k=0;k<WA;k++)
		{
			sum += A[ridA*WA + k]*B[k*WB+cidB];
		}
		C[ridA*WB + cidB] = sum;
	}
}


int main()
{
	int A[100][100],linearA[500],linearB[500],B[100][100],C[100][100],linearC[500];
	int *d_A,*d_B,*d_C,WA,WB,HA,HB,matsizeA,matsizeB,matsizeC,index = 0;
	printf("Enter num of rows and num of columns of matrix A: ");
	scanf("%d %d",&HA,&WA);
	matsizeA = WA*HA*sizeof(int);
	printf("Enter matrix A of size %dx%d:\n",HA,WA);
	for(int i=0;i<HA;i++){
		for(int j=0;j<WA;j++){
			scanf("%d",&A[i][j]);
			linearA[index++] = A[i][j];
		}
	}
	printf("Enter num of rows and num of columns of matrix A: ");
	scanf("%d %d",&HB,&WB);
	matsizeB = WB*HB*sizeof(int);
	printf("Enter matrix B of size %dx%d:\n",HB,WB);
	index = 0;
	for(int i=0;i<HB;i++){
		for(int j=0;j<WB;j++){
			scanf("%d",&B[i][j]);
			linearB[index++] = B[i][j];
		}
	}

	matsizeC = HA*WB*sizeof(int);
	hipMalloc((void**)&d_A,matsizeA);
	hipMalloc((void**)&d_B,matsizeB);
	hipMalloc((void**)&d_C,matsizeC);
	hipMemcpy(d_A,linearA,matsizeA,hipMemcpyHostToDevice);
	hipMemcpy(d_B,linearB,matsizeB,hipMemcpyHostToDevice);

	matrixmulrow<<<1,HA>>>(d_A,d_B,d_C,WB,WA);

	hipMemcpy(linearC,d_C,matsizeC,hipMemcpyDeviceToHost);
	index = 0;
	for(int i=0;i<HA;i++){
		for(int j=0;j<WB;j++){
			C[i][j] = linearC[index++];
		}
	}
	printf("Product of the 2 matrices:\n");
	for(int i=0;i<HA;i++){
		for(int j=0;j<WB;j++){
			printf("%d ",C[i][j]);
		}
		printf("\n");
	}

	matrixmulcol<<<1,WB>>>(d_A,d_B,d_C,HA,WA);

	hipMemcpy(linearC,d_C,matsizeC,hipMemcpyDeviceToHost);
	index = 0;
	for(int i=0;i<HA;i++){
		for(int j=0;j<WB;j++){
			C[i][j] = linearC[index++];
		}
	}
	printf("Product of the 2 matrices:\n");
	for(int i=0;i<HA;i++){
		for(int j=0;j<WB;j++){
			printf("%d ",C[i][j]);
		}
		printf("\n");
	}

	dim3 dimGrid(1,1,1);
	dim3 dimBlock(WB,HA,1);
	matrixaddrc<<<dimGrid,dimBlock>>>(d_A,d_B,d_C,WA);
	hipMemcpy(linearC,d_C,matsizeC,hipMemcpyDeviceToHost);
	index = 0;
	for(int i=0;i<HA;i++){
		for(int j=0;j<WB;j++){
			C[i][j] = linearC[index++];
		}
	}
	printf("Product of the 2 matrices:\n");
	for(int i=0;i<HA;i++){
		for(int j=0;j<WB;j++){
			printf("%d ",C[i][j]);
		}
		printf("\n");
	}
	hipFree(d_A);hipFree(d_B);hipFree(d_C);
}