#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdio.h>

// Define matrix dimensions (M and N)
#define M 4
#define N 4

__global__ void onesComplement(int *A, int *B, int rows, int cols)
{
    int rid = blockIdx.y * blockDim.y + threadIdx.y;
    int cid = blockIdx.x * blockDim.x + threadIdx.x;

    if (rid >= 0 && rid < rows && cid >= 0 && cid < cols)
    {
        if (rid > 0 && rid < rows - 1 && cid > 0 && cid < cols - 1)
        {
            // Calculate the index for the current element
            int index = rid * cols + cid;
            // Calculate the 1's complement of the element and store it in B
            int number = A[index];
            int rev = 0;
            for (int i = 0; number > 0; i++)
            {
                rev *= 10;
                rev += 1 - number % 2;
                number = number / 2;
            }
            B[index] = rev;
        }
        else
        {
            // Copy border elements as-is
            B[rid * cols + cid] = A[rid * cols + cid];
        }
    }
}

int main()
{
    int A[M][N];
    int B[M][N];

    for (int i = 0; i < M; i++){
        for (int j = 0; j < N; j++){
            A[i][j] = i * N + j;
        }
    }

    int *d_A, *d_B; // Device pointers for matrices A and B

    hipMalloc((void **)&d_A, M * N * sizeof(int));
    hipMalloc((void **)&d_B, M * N * sizeof(int));
    hipMemcpy(d_A, A, M * N * sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (M + threadsPerBlock.y - 1) / threadsPerBlock.y);

    onesComplement<<<numBlocks, threadsPerBlock>>>(d_A, d_B, M, N);

    hipMemcpy(B, d_B, M * N * sizeof(int), hipMemcpyDeviceToHost);

    printf("Matrix A:\n");
    for (int i = 0; i < M; i++){
        for (int j = 0; j < N; j++){
            printf("%d ", A[i][j]);
        }
        printf("\n");
    }

    printf("Matrix B (1's complement of non-border elements in binary):\n");
    for (int i = 0; i < M; i++){
        for (int j = 0; j < N; j++){
            printf("%d ", B[i][j]);
        }
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_B);

    return 0;
}