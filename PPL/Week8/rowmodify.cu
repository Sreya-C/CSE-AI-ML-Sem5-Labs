#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void rowmodify(int * A,int width)
{
	int ridA = threadIdx.x + blockIdx.x*blockDim.x;
	int ele;
	for(int cidA = 0;cidA < width;cidA++){
		ele = A[ridA*width + cidA];
		A[ridA*width + cidA] = powf(ele,ridA+1);
	}
}

int main()
{
	int A[100][100],linearA[10000];
	int * d_A, row, col,matsize,index = 0;
	printf("Enter no of rows and columns of matrix A: ");
	scanf("%d %d",&row,&col);
	matsize = row*col*sizeof(int);
	printf("Enter matrix A of size %dx%d:\n",row,col);
	for(int i=0;i<row;i++){
		for(int j=0;j<col;j++){
			scanf("%d",&A[i][j]);
			linearA[index++] = A[i][j];
		}
	}

	hipMalloc((void **)&d_A,matsize);
	hipMemcpy(d_A,linearA,matsize,hipMemcpyHostToDevice);

	rowmodify<<<1,row>>>(d_A,col);

	hipMemcpy(linearA,d_A,matsize,hipMemcpyDeviceToHost);

	printf("Resultant Matrix:\n");
	index = 0;
	for(int i=0;i<row;i++){
		for(int j=0;j<col;j++){
			A[i][j] = linearA[index++];
			printf("%d ",A[i][j]);
		}
		printf("\n");
	}

	hipFree(d_A);
}