#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void spvm(float * data,int num_rows,int * col_index, int * row_ptr, float * x, float * y)
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	float result = 0;
	if(row<num_rows){
		int row_start = row_ptr[row];
		int row_end = row_ptr[row+1];
		for( int ele=row_start;ele<row_end;ele+=1){
			result += data[ele]*x[col_index[ele]];
		}
		y[row] = result;
	}
}

int main()
{
	float A[50][50], * data, * x, * y;
	int * row_ptr, * col_index, row, col, matsize,data_ind=0;
	printf("Enter no of rows and columns of matrix A: ");
	scanf("%d %d",&row,&col);
	matsize = row*col*sizeof(int);

	float * d_data, * d_x, * d_y;
	int * d_col_index, * d_row_ptr;

	data = (float *) malloc(matsize*sizeof(float));
	x = (float *) malloc(row*sizeof(float));
	y = (float *) malloc(row*sizeof(float));
	row_ptr = (int *) malloc((row+1)*sizeof(int));
	col_index = (int *) malloc(matsize*sizeof(int));
	row_ptr[0] = 0;

	printf("Enter sparse matrix A of size %dx%d:\n",row,col);
	for(int i=0;i<row;i++){
		for(int j=0;j<col;j++){
			scanf("%f",&A[i][j]);
			if(A[i][j] >0){
				col_index[data_ind] = j;  
				data[data_ind++]= A[i][j];
			}
		}
		row_ptr[i+1] = data_ind; 
	}
	printf("Data: ");
	for(int i=0;i<data_ind;i++){
		printf("%f ",data[i]);
	}
	printf("\nColumn_Index: ");
	for(int i=0;i<data_ind;i++){
		printf("%d ",col_index[i]);
	}
	printf("\nRow_Ptr: ");
	for(int i=0;i<row+1;i++){
		printf("%d ",row_ptr[i]);
	}

	printf("\nEnter %d elements of x: ", col);
	for(int i=0;i<row;i++){
		scanf("%f", &x[i]);
		printf("%f ",x[i]);
	}

	hipMalloc((void **)&d_data,(data_ind+1)*sizeof(float));
	hipMalloc((void **)&d_x,(row)*sizeof(float));
	hipMalloc((void **)&d_y,(row)*sizeof(float));
	hipMalloc((void **)&d_row_ptr, (row+1)*sizeof(int));
	hipMalloc((void **)&d_col_index, (data_ind)*sizeof(int));

	hipMemcpy(d_data,data,(data_ind+1)*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_x,x,(row)*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_row_ptr,row_ptr,(row+1)*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_col_index,col_index,(data_ind)*sizeof(int),hipMemcpyHostToDevice);

	spvm<<<ceil(row/256.0), 256>>>(d_data,row,d_col_index,d_row_ptr,d_x,d_y);

	hipMemcpy(y,d_y,(row)*sizeof(float),hipMemcpyDeviceToHost);

	printf("\nFinal Vector:\n");
	for(int i=0;i<row;i++){
		printf("%f\n",y[i]);
	}

	hipFree(d_data);	hipFree(d_col_index);	hipFree(d_y);	hipFree(d_x);

}

