#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void convolution(int *N, int *M, int *P, int width, int mask_width)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int Pvalue = 0;
    int N_start_point = tid - (mask_width / 2);
    for (int j = 0; j < mask_width; j++)
    {
        if (N_start_point + j >= 0 && N_start_point + j < width)
        {
            Pvalue += N[N_start_point + j] * M[j];
        }
    }
    P[tid] = Pvalue;
}

int main(void)
{

    int width, mask_width;

    printf("Enter width of array: ");
    scanf("%d", &width);
    printf("Enter width of mask: ");
    scanf("%d", &mask_width);
    int *N = (int *)malloc(width * sizeof(int));
    int *M = (int *)malloc(mask_width * sizeof(int));
    int *P = (int *)malloc(width * sizeof(int));
    printf("Enter values of array: ");
    for (int i = 0; i < width; i++)
    {
        scanf("%d", &N[i]);
    }

    printf("Enter values of mask: ");
    for (int i = 0; i < mask_width; i++)
    {
        scanf("%d", &M[i]);
    }
    int *d_N, *d_M, *d_P;
    hipMalloc((void **)&d_N, width * sizeof(int));
    hipMalloc((void **)&d_M, mask_width * sizeof(int));
    hipMalloc((void **)&d_P, width * sizeof(int));
    hipMemcpy(d_N, N, width * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_M, M, mask_width * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_P, P, width * sizeof(int), hipMemcpyHostToDevice);
    int block_size = 256;
    int grid_size = (width + block_size - 1) / block_size;
    convolution<<<grid_size, block_size>>>(d_N, d_M,d_P, width, mask_width);

    hipMemcpy(P, d_P, width * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < width; i++)
    {
        printf("%d ", P[i]);
    }
    printf("\n");
    hipFree(d_N);
    hipFree(d_M);
    hipFree(d_P);

    return 0;
}