
#include <hip/hip_runtime.h>
#include <stdio.h>

// Define matrix dimensions and kernel dimensions
#define WIDTH 4
#define HEIGHT 4
#define KERNEL_SIZE 3

// CUDA kernel for 2D convolution
__global__ void convolution2D(int *input, int *mask, int *output, int width, int height, int kernelSize)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < height && col < width)
    {
        int sum = 0;
        int offset = kernelSize / 2;
        for (int i = 0; i < kernelSize; i++)
        {
            for (int j = 0; j < kernelSize; j++)
            {
                int r = row + i - offset;
                int c = col + j - offset;

                if (r >= 0 && r < height && c >= 0 && c < width)
                {
                    sum += input[r * width + c] * mask[i * kernelSize + j];
                }
            }
        }

        output[row * width + col] = sum;
    }
}

int main()
{
    int input[HEIGHT][WIDTH];                                               
    int mask[KERNEL_SIZE][KERNEL_SIZE] = {{1, 1, 1}, {1, 1, 1}, {1, 1, 1}}; 
    int output[HEIGHT][WIDTH];                                                                                                                                                   

    int *d_input, *d_mask, *d_output;

    // Initialize input matrix (for simplicity)
    for (int i = 0; i < HEIGHT; i++)
    {
        for (int j = 0; j < WIDTH; j++)
        {
            input[i][j] = 1;
        }
    }
 
    // Allocate memory on the device
    hipMalloc((void **)&d_input, WIDTH * HEIGHT * sizeof(int));
    hipMalloc((void **)&d_mask, KERNEL_SIZE * KERNEL_SIZE * sizeof(int));
    hipMalloc((void **)&d_output, WIDTH * HEIGHT * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_input, input, WIDTH * HEIGHT * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, KERNEL_SIZE * KERNEL_SIZE * sizeof(int), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 dimGrid((WIDTH + 15) / 16, (HEIGHT + 15) / 16);
    dim3 dimBlock(16, 16);

    // Launch the CUDA kernel
    convolution2D<<<dimGrid, dimBlock>>>(d_input, d_mask, d_output, WIDTH, HEIGHT, KERNEL_SIZE);

    // Copy the result back to the host
    hipMemcpy(output, d_output, WIDTH * HEIGHT * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_mask);
    hipFree(d_output);

    printf("Input Matrix:\n");
    // Print the input matrix
    for (int i = 0; i < HEIGHT; i++)
    {
        for (int j = 0; j < WIDTH; j++)
        {
            printf("%d\t", input[i][j]);
        }
        printf("\n");
    }

    printf("Mask:\n");
    // Print the input matrix
    for (int i = 0; i < KERNEL_SIZE; i++)
    {
        for (int j = 0; j < KERNEL_SIZE; j++)
        {
            printf("%d\t", mask[i][j]);
        }
        printf("\n");
    }

    printf("Output Matrix:\n");
    // Print the output matrix
    for (int i = 0; i < HEIGHT; i++)
    {
        for (int j = 0; j < WIDTH; j++)
        {
            printf("%d\t", output[i][j]);
        }
        printf("\n");
    }
    return 0;
}