#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#define STB_IMAGE_IMPLEMENTATION
#include "./stb_image.h"
#define STBI_MSC_SECURE_CRT
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "./stb_image_write.h"

#include <stdint.h>

#define BLOCK_WIDTH 32

__global__ void rgbToGray(unsigned char* img_in, unsigned char* img_out, int height, int width) {
    int rid = blockIdx.y * blockDim.y + threadIdx.y;
    int cid = blockIdx.x * blockDim.x + threadIdx.x;

    if(rid < height && cid < width) {
        int grayOffset = rid * width + cid;
        int rgbOffset = grayOffset * 3;
        unsigned char r=img_in[rgbOffset], g=img_in[rgbOffset+1], b=img_in[rgbOffset+2];
        img_out[rid*width+cid] = (unsigned char) (0.21f * r + 0.71f * g + 0.07f * b);
    }
}

__global__ void emboss(unsigned char* img_in, int* img_out, int height, int width, int* min, int* max) {
    int rid = blockIdx.y * blockDim.y + threadIdx.y;
    int cid = blockIdx.x * blockDim.x + threadIdx.x;

    if(rid < height && cid < width) {
        int gradX = 0, gradY = 0;
        
        if(cid - 1 >= 0) 
            gradX += img_in[rid*width+(cid-1)];
        if(cid + 1 < width)
            gradX -= img_in[rid*width+(cid+1)];
        
        if(rid - 1 >= 0) 
            gradY += img_in[(rid-1)*width+cid];
        if(rid + 1 < height)
            gradY -= img_in[(rid+1)*width+cid];
        
        int val = gradX + gradY;
        
        img_out[rid*width+cid] = val;
        atomicMin(min, val);
        atomicMax(max, val);
    }
}

__global__ void normalize(int* img_in, unsigned char* img_out, int height, int width, int min, int max) {
    int rid = blockIdx.y * blockDim.y + threadIdx.y;
    int cid = blockIdx.x * blockDim.x + threadIdx.x;

    if(rid < height && cid < width) {
        int offSet = rid * width + cid;
        img_out[offSet] = (unsigned char) ((img_in[offSet] - min) * 255 / (max - min));
    }
}


int main() {
    unsigned char *img_in, *img_out;
    int width, height, bpp, min=INT_MAX, max=INT_MIN;
    int sizeimgin, sizeimggray, sizeimgint, sizeimgout;

    unsigned char *d_img_in, *d_img_gray, *d_img_out;
    int *d_img_int, *d_min, *d_max; 

    img_in = stbi_load("lena.jpeg", &width, &height, &bpp, 0);

    sizeimgin = width * height * bpp * sizeof(unsigned char);
    sizeimggray = width * height * 1 * sizeof(unsigned char);
    sizeimgint = width * height * 1 * sizeof(int);
    sizeimgout = width * height * 1 * sizeof(unsigned char);

    img_out = (unsigned char*) malloc(sizeimgout);

    hipMalloc((void**) &d_img_in, sizeimgin);
    hipMalloc((void**) &d_img_gray, sizeimggray);
    hipMalloc((void**) &d_img_int, sizeimgint);
    hipMalloc((void**) &d_min, sizeof(int));
    hipMalloc((void**) &d_max, sizeof(int));
    hipMalloc((void**) &d_img_out, sizeimgout);

    hipMemcpy(d_img_in, img_in, sizeimgin, hipMemcpyHostToDevice);
    hipMemcpy(d_min, &min, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_max, &max, sizeof(int), hipMemcpyHostToDevice);
    
    dim3 gridDim(ceil((float)width/BLOCK_WIDTH), ceil((float)height/BLOCK_WIDTH));
    dim3 blockDim(BLOCK_WIDTH, BLOCK_WIDTH);

    rgbToGray<<<gridDim, blockDim>>>(d_img_in, d_img_gray, height, width);

    emboss<<<gridDim, blockDim>>>(d_img_gray, d_img_int, height, width, d_min, d_max);

    hipMemcpy(&min, d_min, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&max, d_max, sizeof(int), hipMemcpyDeviceToHost);

    normalize<<<gridDim, blockDim>>>(d_img_int, d_img_out, height, width, min, max);

    hipMemcpy(img_out, d_img_out, sizeimgout, hipMemcpyDeviceToHost);

    stbi_write_jpg("lena_emboss.jpg", width, height, 1, img_out, 100);

    hipFree(d_img_in);
    hipFree(d_img_gray);
    hipFree(d_img_int);
    hipFree(d_min);
    hipFree(d_max);
    hipFree(d_img_out);

    free(img_in);
    free(img_out);

    return 0;
}