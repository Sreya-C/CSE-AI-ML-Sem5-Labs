#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>	
#include<stdlib.h>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image/stb_image.h"
#define STBI_MSC_SECURE_CRT
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image/stb_image_write.h"
#include <stdint.h>
#define THREADS 32
__global__ void rgbToGray(unsigned char *in,unsigned char *out,int width,int height)
{
	int Col = blockDim.x*blockIdx.x+threadIdx.x;
	int Row = blockDim.y * blockIdx.y + threadIdx.y;
	if (Col < width && Row < height)
	{
		int grayOffset = Row * width + Col;
		int rgbOffset = grayOffset * 3;
		unsigned char r = in[rgbOffset];
		unsigned char g = in[rgbOffset + 1];
		unsigned char b = in[rgbOffset + 2];
		out[grayOffset] =(unsigned char) (0.21f * r + 0.71f * g + 0.07f * b);
	}
}
int main() {
	unsigned char* h_N;
	unsigned char* d_N;
	unsigned char* d_out;
	hipError_t cudaStatus;
	int width, height, bpp;
	unsigned char* rgb_image = stbi_load("Lena.jpeg", &width, &height, &bpp, 0);
	h_N = (unsigned char*)malloc(width * height * bpp * sizeof(unsigned char));
	h_N = rgb_image;
	unsigned char* h_out= (unsigned char*)malloc(width * height * bpp * sizeof(unsigned char));
	cudaStatus = hipMalloc((void**)&d_N, sizeof(unsigned char) * width*height*bpp);
	printf("%d", cudaStatus);
	cudaStatus = hipMalloc((void**)&d_out, sizeof(unsigned char) * width *height*bpp);
	printf("%d", cudaStatus);
	cudaStatus = hipMemcpy(d_N, h_N, width*height*bpp * sizeof(unsigned char), hipMemcpyHostToDevice);
	printf("%d", cudaStatus);
	dim3 dimBlock(THREADS, THREADS);
	dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y);
	rgbToGray << < dimGrid,dimBlock >> > (d_N, d_out, width, height);
	hipDeviceSynchronize();
	cudaStatus = hipMemcpy(h_out, d_out, width*height, hipMemcpyDeviceToHost);
	printf("%d", cudaStatus);
	stbi_write_jpg("Lena_gray.jpg", width, height, 1, h_out, width*3);
	hipFree(d_N);
	hipFree(d_out);
	return 0;
}


