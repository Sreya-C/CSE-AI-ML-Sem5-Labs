#include <stdio.h>  
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define N 3

__global__ void matrixMul(int *a, int *b, int *c, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if (row < n && col < n) {
        for (int i = 0; i < n; i++) {
            sum += a[row * n + i] * b[i * n + col];
        }
        c[row * n + col] = sum;
    }
}

int main(void) {
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N * N * sizeof(int);

    // Allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Setup input values
    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    for (int i = 0; i < N * N; i++) {
        a[i] = i;
        b[i] = i+1;
    }

    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Launch add() kernel on GPU
    dim3 dimGrid(1, 1, 1);
    dim3 dimBlock(N, N, 1);
    matrixMul<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, N);

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Print result
    printf("Matrix A:\n");
    for (int i = 0; i < N * N; i++) {
        printf("%d ", a[i]);
        if ((i + 1) % N == 0) {
            printf("\n");
        }
    }
    printf("\n");

    printf("Matrix B:\n");
    for (int i = 0; i < N * N; i++) {
        printf("%d ", b[i]);
        if ((i + 1) % N == 0) {
            printf("\n");
        }
    }
    printf("\n");

    printf("Matrix C:\n");
    for (int i = 0; i < N * N; i++) {
        printf("%d ", c[i]);
        if ((i + 1) % N == 0) {
            printf("\n");
        }
    }
    printf("\n");

    // Cleanup
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}