#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#define BLOCK_WIDTH 2
#define TILE_WIDTH 2
#define WIDTH 4

__device__ int getTid() {
	int blockSkip = (blockIdx.y * gridDim.x * blockDim.x * blockDim.y);
	int rowSkip = (threadIdx.y * gridDim.x * blockDim.x);
	int rowDisplacement = (blockIdx.x * blockDim.x) + threadIdx.x;
	int tid = blockSkip + rowSkip + rowDisplacement;
	printf("Currently in block x %d, block y %d, thread x %d, thread x %d,blockSkip = %d,rowSkip = %d,rowDisplacement = %d\n", blockIdx.x,blockIdx.y,threadIdx.x,threadIdx.y,blockSkip,rowSkip,rowDisplacement);
	return tid;
}

__global__ void MatAddElementThread(int *a, int *b, int *d) {
	int tid = getTid();
	d[tid] = a[tid] + b[tid];
}

int main() 
{
	int *matA, *matB, *matSum;
	int *da, *db, *dc;
	printf("\n== Enter elements of Matrix A (4x4) ==\n");
	matA = (int*)malloc(sizeof(int) * WIDTH * WIDTH);
	for(int i = 0; i < WIDTH * WIDTH; i++)
	{
		scanf("%d", &matA[i]);
	}
	printf("\n== Enter elements of Matrix B (4x4) ==\n");
	matB = (int*)malloc(sizeof(int) * WIDTH * WIDTH);
	for(int i = 0; i < WIDTH * WIDTH; i++)
	{
		scanf("%d", &matB[i]);
	}

	matSum = (int*)malloc(sizeof(int) * WIDTH * WIDTH);
	hipMalloc((void **) &da, sizeof(int) * WIDTH * WIDTH);
	hipMalloc((void **) &db, sizeof(int) * WIDTH * WIDTH);
	hipMalloc((void **) &dc, sizeof(int) * WIDTH * WIDTH);
	hipMemcpy(da, matA, sizeof(int) * WIDTH *WIDTH, hipMemcpyHostToDevice);
	hipMemcpy(db, matB, sizeof(int) * WIDTH *WIDTH, hipMemcpyHostToDevice);
	int NumBlocks = WIDTH / BLOCK_WIDTH;
	dim3 grid_conf (NumBlocks, NumBlocks);
	dim3 block_conf (BLOCK_WIDTH, BLOCK_WIDTH);

	MatAddElementThread<<<grid_conf, block_conf>>>(da, db, dc);

	hipMemcpy(matSum,dc,sizeof(int)* WIDTH *WIDTH,hipMemcpyDeviceToHost);
	printf("\n-=Result of Addition=-\n");
	printf("\n");	
	for (int i = 0; i < WIDTH; i++ ) {
		for (int j = 0; j < WIDTH; j++) {
			printf("%6d ", matSum[i * WIDTH + j]);}
		printf("\n");
	}

	hipFree(da);
	hipFree(db);
	hipFree(dc);
	free(matA);
	free(matB);
	free(matSum);
	return 0;
}